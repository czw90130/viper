#include "hip/hip_runtime.h"
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     https://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cfloat>
#include <fstream>
#include <iostream>
#include <thread>

// 定义tiny object loader的实现
#define TINYOBJLOADER_IMPLEMENTATION
// 使用PNG格式
#define USE_PNG

// 引入OpenGP的库头文件
#include <OpenGP/GL/Application.h>
#include <OpenGP/GL/Components/GUICanvasComponent.h>
#include <OpenGP/GL/ImguiRenderer.h>
#include <OpenGP/Image/Image.h>

// 引入本地的头文件
#include "OctopusComponent.h"
#include "Scene.h"

// 引入CUDA头文件来处理碰撞
#include "CollisionGrid.cuh"

// 定义所有OpenGP的实现在这个文件
#define OPENGP_IMPLEMENT_ALL_IN_THIS_FILE
#include <OpenGP/util/implementations.h>

// 使用OpenGP的命名空间
using namespace OpenGP;

int main(int argc, char **argv) {

    // 定义阴影的大小
    int shadow_size = 2048;

    // 初始化应用
    Application app;

    // 初始化场景
    Scene scene;

    // 创建一个实体作为灯源，并附带一个摄像机组件
    auto &light_entity = scene.create_entity_with<CameraComponent>();
    // 设置灯源的方向
    light_entity.get<TransformComponent>().set_forward(
        Vec3(-1, -2, 0).normalized());
    // 设置灯源的位置
    light_entity.get<TransformComponent>().position = Vec3(50, 100, 0);

    // 计算投影和视图矩阵，用于计算阴影
    Mat4x4 shadow_matrix =
        (light_entity.get_projection(shadow_size, shadow_size) *
         light_entity.get_view());

    // 创建一个用于渲染的地板实体
    auto &floor_entity = scene.create_entity_with<WorldRenderComponent>();
    // 设置地板的渲染器
    auto &floor_renderer = floor_entity.set_renderer<SurfaceMeshRenderer>();
    floor_renderer.get_gpu_mesh().set_vpoint(
        {Vec3(-10000, 0, -10000), Vec3(10000, 0, -10000),
         Vec3(-10000, 0, 10000), Vec3(10000, 0, 10000)});
    floor_renderer.get_gpu_mesh().set_vnormal(
        {Vec3(0, 1, 0), Vec3(0, 1, 0), Vec3(0, 1, 0), Vec3(0, 1, 0)});
    floor_renderer.get_gpu_mesh().set_triangles({0, 1, 2, 1, 2, 3});

    Material floormat(R"GLSL(

        uniform sampler2D shadow_map;

        uniform vec3 light_pos;
        uniform mat4 shadow_matrix;
        uniform float shadow_near;
        uniform float shadow_far;

        vec3 world2uvdepth(vec3 pos, mat4 mat) {
            vec4 a = mat * vec4(pos, 1);
            vec3 b = a.xyz / a.w;
            return (b + vec3(1)) / 2;
        }

        float get_shadow_mask(vec2 uv) {
            return 1 - smoothstep(0.3, 0.5, length(uv - vec2(0.5, 0.5)));
        }

        vec3 get_ambient(vec3 pos) {
            vec3 ambient = vec3(0.14, 0.14, 0.18);

            vec3 uvd = world2uvdepth(pos, shadow_matrix);

            return ambient + vec3(0.2) * get_shadow_mask(uvd.xy);
        }

        float linear_shadow_depth(float d) {
            return shadow_near * shadow_far / (shadow_far + d * (shadow_near - shadow_far));
        }

        float get_shadow(vec3 pos) {
            ivec2 dim = textureSize(shadow_map, 0);
            vec3 uvd = world2uvdepth(pos, shadow_matrix);

            vec2 base_coord = uvd.xy * dim;
            ivec2 base_coord_i = ivec2(floor(base_coord));
            vec2 inter = fract(base_coord);

            mat4 shadow_depths;
            for (int i = 0; i < 4; ++i) {
                for (int j = 0; j < 4; ++j) {
                    shadow_depths[i][j] = linear_shadow_depth(texelFetch(shadow_map, base_coord_i + ivec2(i-1, j-1), 0).r);
                }
            }

            float threshold = linear_shadow_depth(uvd.z) - 0.1;

            mat2 pcf_vals = mat2(0);
            for (int i = 0; i < 2; ++i) {
                for (int j = 0; j < 2; ++j) {
                    for (int x = 0; x < 3; ++x) {
                        for (int y = 0; y < 3; ++y) {
                            pcf_vals[i][j] += (shadow_depths[x + i][y + j] < threshold) ? 0 : (1.0 / 9.0);
                        }
                    }
                }
            }

            float a = mix(pcf_vals[0][0], pcf_vals[1][0], inter.x);
            float b = mix(pcf_vals[0][1], pcf_vals[1][1], inter.x);

            return mix(a, b, inter.y) * get_shadow_mask(uvd.xy);
        }

        vec4 fragment_shade() {
            vec3 pos = get_position();

            vec3 lightdir = normalize(light_pos - pos);

            vec3 white_color = vec3(1, 1, 1);
            vec3 black_color = vec3(0.6, 0.6, 0.6);

            vec3 background = (white_color + black_color) / 2;

            vec3 diffuse_color = white_color;

            vec3 modpos = mod(pos / 5, 1);

            if ((modpos.x < 0.5) ^^ (modpos.z < 0.5)) {
                diffuse_color = black_color;
            }

            float blur = exp(-2 * max(length(dFdx(pos)), length(dFdy(pos))));
            blur = clamp(2 * blur, 0, 1);

            diffuse_color = mix(background, diffuse_color, blur);

            vec3 ambient = get_ambient(pos);

            float shadow = get_shadow(pos);

            vec3 out_color = shadow * 0.85 * clamp(dot(get_normal(), normalize(lightdir)), 0, 1) * diffuse_color;

            out_color += ambient * diffuse_color;

            return vec4(out_color, 1);
        }

    )GLSL");
    floormat.set_property("ao_map", 6);
    floormat.set_property("shadow_map", 7);
    floormat.set_property("shadow_matrix", shadow_matrix);
    floormat.set_property("light_pos",
                          light_entity.get<TransformComponent>().position);
    floormat.set_property("shadow_near", light_entity.near_plane);
    floormat.set_property("shadow_far", light_entity.far_plane);
    floor_renderer.set_material(floormat);
    floor_renderer.rebuild();

    // 创建一个模拟场景  
    viper::Scene sim_scene;

    // 为OctopusComponent设置模拟场景
    OctopusComponent::v_scene = &sim_scene;
    auto &octoswarm = scene.create_entity_with<OctopusComponent>();

    octoswarm.renderer->get_material().set_property("shadow_matrix",
                                                    shadow_matrix);
    octoswarm.renderer->get_material().set_property(
        "light_pos", light_entity.get<TransformComponent>().position);
    octoswarm.renderer->get_material().set_property("shadow_near",
                                                    light_entity.near_plane);
    octoswarm.renderer->get_material().set_property("shadow_far",
                                                    light_entity.far_plane);
    octoswarm.sphere_renderer->get_material().set_property("shadow_matrix",
                                                           shadow_matrix);
    octoswarm.sphere_renderer->get_material().set_property(
        "light_pos", light_entity.get<TransformComponent>().position);
    octoswarm.sphere_renderer->get_material().set_property(
        "shadow_near", light_entity.near_plane);
    octoswarm.sphere_renderer->get_material().set_property(
        "shadow_far", light_entity.far_plane);
    octoswarm.tsphere_renderer->get_material().set_property("shadow_matrix",
                                                            shadow_matrix);
    octoswarm.tsphere_renderer->get_material().set_property(
        "light_pos", light_entity.get<TransformComponent>().position);
    octoswarm.tsphere_renderer->get_material().set_property(
        "shadow_near", light_entity.near_plane);
    octoswarm.tsphere_renderer->get_material().set_property(
        "shadow_far", light_entity.far_plane);
    octoswarm.cannonball_renderer->get_material().set_property("shadow_matrix",
                                                               shadow_matrix);
    octoswarm.cannonball_renderer->get_material().set_property(
        "light_pos", light_entity.get<TransformComponent>().position);
    octoswarm.cannonball_renderer->get_material().set_property(
        "shadow_near", light_entity.near_plane);
    octoswarm.cannonball_renderer->get_material().set_property(
        "shadow_far", light_entity.far_plane);
    octoswarm.pillar_renderer->get_material().set_property("shadow_matrix",
                                                           shadow_matrix);
    octoswarm.pillar_renderer->get_material().set_property(
        "light_pos", light_entity.get<TransformComponent>().position);
    octoswarm.pillar_renderer->get_material().set_property(
        "shadow_near", light_entity.near_plane);
    octoswarm.pillar_renderer->get_material().set_property(
        "shadow_far", light_entity.far_plane);

    // 创建一个带有Trackball组件的实体
    auto &c_entity = scene.create_entity_with<TrackballComponent>();
    c_entity.oriented = true;

    // 设置窗口的宽和高
    int ww = 3840, wh = 1080;

    // 以下是创建和管理Framebuffer，纹理等的代码

    Framebuffer fb, fb_shadow;
    RGB8Texture color_map, color_map_shadow;
    D32FTexture depth_map, depth_map_shadow;

    auto realloc = [&](int w, int h) {
        color_map.allocate(w, h);
        depth_map.allocate(w, h);
    };

    realloc(ww, wh);

    depth_map_shadow.allocate(shadow_size, shadow_size);
    color_map_shadow.allocate(shadow_size, shadow_size);

    fb.attach_color_texture(color_map);
    fb.attach_depth_texture(depth_map);

    fb_shadow.attach_color_texture(color_map_shadow);
    fb_shadow.attach_depth_texture(depth_map_shadow);

    RGB8Texture colmap;
    Image<Eigen::Matrix<uint8_t, 3, 1>> colmap_cpu(2048, 2048);
    std::ifstream("texture.bin", std::ios::binary).read(
        reinterpret_cast<char*>(&colmap_cpu(0, 0)), 12582912);
    colmap.upload(colmap_cpu);

    // 创建一个全屏四边形
    FullscreenQuad fsquad;

    bool show_pills = false;
    bool splitscreen = false;

    auto set_pill_visibility = [&](bool visible) {
        show_pills = visible;
        octoswarm.render_comp->visible = !visible;
        octoswarm.sphere_render_comp->visible = visible;
        octoswarm.vis_update();
    };

    auto draw_scene = [&](int width, int height, int x, int y) {
        //======================================================================
        // Draw shadow map

        fb_shadow.bind();

        light_entity.draw(shadow_size, shadow_size);

        fb_shadow.unbind();

        //======================================================================
        // Draw scene with shadows

        fb.bind();

        glActiveTexture(GL_TEXTURE5);
        colmap.bind();
        glActiveTexture(GL_TEXTURE7);
        depth_map_shadow.bind();

        glActiveTexture(GL_TEXTURE0);

        auto &cam = c_entity.get<CameraComponent>();

        cam.draw(color_map.get_width(), color_map.get_height(), 0, 0, false);

        if (octoswarm.sphere_render_comp->visible) {
            RenderContext context;

            glDepthMask(GL_FALSE);

            context.aspect =
                (float)color_map.get_width() / (float)color_map.get_height();
            context.vfov = cam.vfov;
            context.near = cam.near_plane;
            context.far = cam.far_plane;
            context.eye = cam.get<TransformComponent>().position;
            context.forward = cam.get<TransformComponent>().forward();
            context.up = cam.get<TransformComponent>().up();

            context.update_view();
            context.update_projection();

            auto &renderable = *octoswarm.tsphere_render_comp;
            auto &transform = renderable.get<TransformComponent>();

            context.translation = transform.position;
            context.scale = transform.scale;
            context.rotation = transform.rotation;

            context.update_model();

            glEnable(GL_DEPTH_TEST);
            renderable.get_renderer().render(context);

            glDepthMask(GL_TRUE);
        }

        cam.draw_gui();

        fb.unbind();

        //======================================================================
        // Draw color map to window

        glViewport(x, y, width, height);
        fsquad.draw_texture(color_map);
    };

    // 创建一个窗口
    auto &window = app.create_window([&](Window &window) {
        std::tie(ww, wh) = window.get_size();

        int fbw_new = splitscreen ? ww / 2 : ww;
        int fbh_new = wh;

        int fbw = color_map.get_width();
        int fbh = color_map.get_height();

        if (fbw_new != fbw || fbh_new != fbh) {
            realloc(fbw_new, fbh_new);
        }

        if (splitscreen) {
            set_pill_visibility(false);
            draw_scene(ww / 2, wh, 0, 0);
            set_pill_visibility(true);
            draw_scene(ww / 2, wh, ww / 2, 0);
        } else {
            octoswarm.vis_update();
            draw_scene(ww, wh, 0, 0);
        }
    });

    // 设置窗口大小和标题
    window.set_size(ww, wh);
    window.set_title("VIPER Demo");

    // 以下是关于输入处理和UI的代码
    auto &input = window.get_input();

    c_entity.get<CameraComponent>().set_window(window);
    c_entity.center = Vec3(0, 1, 0);
    c_entity.get<TransformComponent>().position = Vec3(-12, 1, 0);

    auto &bsphere_entity = scene.create_entity_with<WorldRenderComponent>();
    auto &bsphere_renderer = bsphere_entity.set_renderer<SphereMeshRenderer>();

    auto get_mouse_ray = [&](Vec3 &eye, Vec3 &dir) {
        Vec2 pos = input.mouse_position;
        pos[1] = wh - pos[1];

        int w = splitscreen ? ww / 2 : ww;
        pos = 2 * pos.cwiseQuotient(Vec2(w, wh)) - Vec2(1, 1);

        Vec4 cs(pos[0], pos[1], 0.1, 1);

        auto &cam = c_entity.get<CameraComponent>();
        Mat4x4 inv_mat = (cam.get_projection(w, wh) * cam.get_view()).inverse();

        Vec4 world = inv_mat * cs;
        Vec3 p = world.head<3>() / world[3];

        eye = c_entity.get<TransformComponent>().position;
        dir = (p - eye).normalized();
    };

    int framerate = 0;
    double frametime = 0;
    double sim_frametime = 0;

    float playback = 1.0;

    int it_count = 10;

    bool hide_gui = false;
    bool simulating = true;
    bool single_step = false;
    bool bsphere_vis = false;

    std::vector<float> framerates(120);

    auto set_defaults = [&]() {
        show_pills = false;
        octoswarm.render_comp->visible = !show_pills;
        octoswarm.sphere_render_comp->visible = show_pills;
        it_count = 10;
        sim_scene.gravity_strength = 1.0;
        playback = 1.0;
    };

    set_defaults();

    auto &canvas = scene.create_entity_with<GUICanvasComponent>();
    canvas.set_action([&]() {
        if (hide_gui)
            return;

        ImGui::SetNextWindowSize(ImVec2(400, 500));

        ImGui::Begin("Controls", nullptr,
                     ImGuiWindowFlags_NoResize |
                         ImGuiWindowFlags_NoSavedSettings);

        char fr_label[256];
        sprintf(fr_label,
                "Framerate %i fps\n Total:   %3.1f ms\n Sim:     %3.1f ms",
                framerate, frametime, sim_frametime);

        ImGui::PlotLines(fr_label, &(framerates[0]), framerates.size(), 0, "",
                         0, 60);

        ImGui::Separator();

        if (ImGui::Button("Reset")) {
            octoswarm.reset();
        }

        ImGui::SameLine(0, 4);

        const char *bname = simulating ? "Pause" : "Resume";
        if (ImGui::Button(bname)) {
            simulating = !simulating;
        }

        if (!simulating) {
            ImGui::SameLine(0, 4);
            if (ImGui::Button("Step")) {
                single_step = true;
            }
        }

        ImGui::Checkbox("Split Screen", &splitscreen);

        if (ImGui::Checkbox("Show Primitives", &show_pills)) {
            set_pill_visibility(show_pills);
        }

        ImGui::SliderFloat("Gravity", &sim_scene.gravity_strength, -1.0f, 3.0f);
        ImGui::SliderInt("Solver Iterations", &it_count, 0, 50);

        if (ImGui::Button("Set Defaults")) {
            set_defaults();
            octoswarm.vis_update();
        }

        ImGui::Separator();

        const char *const scenes[] = {"Empty", "Pillars", "Cannonballs",
                                      "Explosion"};

        if (ImGui::ListBox("Scenes", &octoswarm.scene_index, scenes,
                           sizeof(scenes) / sizeof(scenes[0]))) {
            octoswarm.reset();
        }

        ImGui::Separator();

        ImGui::LabelText("Controls", "Look:                   Middle Mouse");
        ImGui::Text("Recenter:                Right Mouse");
        ImGui::Text("Pan:            Shift + Middle Mouse");
        ImGui::Text("Grab:                     Left Mouse");
        ImGui::Text("Shoot:                      Spacebar");
        ImGui::Text("Toggle Primitives:               F10");
        ImGui::Text("Pause/Resume:                    F11");
        ImGui::Text("Show/Hide Window:                F12");

        ImGui::End();
    });

    canvas.set_camera(c_entity.get<CameraComponent>());

    int chambered_cow = 0;

    long frame = 0;
    long sim_frame = 0;

    double last_time = glfwGetTime();
    double frame_avg = 0;
    double sim_frame_avg = 0;

    int held = 0;
    int selected = -1;

    bool swapped_pills = false;
    bool swapped_pause = false;
    bool swapped_window = false;
    bool recentered = false;

    app.add_listener<ApplicationUpdateEvent>(
        [&](const ApplicationUpdateEvent &) {
            SphereMesh temp_smesh;
            auto vs_temp =
                temp_smesh.add_vertex(viper::CollisionGrid::b_sphere);
            temp_smesh.add_sphere(vs_temp);
            bsphere_entity.visible = bsphere_vis;
            bsphere_renderer.upload_mesh(temp_smesh);

            if (input.get_mouse(0)) {
                Vec3 eye, dir;
                get_mouse_ray(eye, dir);

                if (selected == -1) {
                    selected = octoswarm.intersect(eye, dir);
                    sim_scene.state.xa[selected] = 0;
                } else {

                    Vec3 p = sim_scene.state.x[selected];

                    Vec3 x = p - eye;
                    Vec3 new_pos = p - (x - dir * dir.dot(x));
                    new_pos[1] =
                        std::max(new_pos[1], sim_scene.state.r[selected]);
                    sim_scene.state.x[selected] = new_pos;
                    sim_scene.state.xp[selected] = sim_scene.state.x[selected];
                }
            } else if (selected != -1) {
                sim_scene.state.xa[selected] = 1;
                selected = -1;
            }

            if (input.get_mouse(1)) {
                Image<float> depth_im;
                depth_map.download(depth_im);

                int mxi = int(input.mouse_position[0]);
                int myi = int(wh - input.mouse_position[1]);

                auto &cam = c_entity.get<CameraComponent>();
                Mat4x4 inv_mat =
                    (cam.get_projection() * cam.get_view()).inverse();

                if (!recentered &&
                    !(mxi < 0 || mxi >= ww || myi < 0 || myi >= wh)) {

                    Vec3 uvdepth;
                    uvdepth.head<2>() = Vec2(float(mxi) / ww, float(myi) / wh);
                    uvdepth[2] = min(depth_im(myi, mxi), 0.999);

                    Vec4 dev(0, 0, 0, 1);
                    dev.head<3>() = 2 * uvdepth - Vec3::Ones();
                    Vec4 world_h = inv_mat * dev;

                    Vec3 new_center = world_h.head<3>() / world_h[3];
                    Vec3 dc = new_center - c_entity.center;
                    c_entity.center += dc;
                    c_entity.get<TransformComponent>().position += dc;

                    recentered = true;
                }
            } else {
                recentered = false;
            }

            if (input.get_key(GLFW_KEY_F10)) {
                if (!swapped_pills) {
                    set_pill_visibility(!show_pills);
                    swapped_pills = true;
                }
            } else {
                swapped_pills = false;
            }
            if (input.get_key(GLFW_KEY_F11)) {
                if (!swapped_pause) {
                    simulating = !simulating;
                    swapped_pause = true;
                }
            } else {
                swapped_pause = false;
            }
            if (input.get_key(GLFW_KEY_F12)) {
                if (!swapped_window) {
                    hide_gui = !hide_gui;
                    swapped_window = true;
                }
            } else {
                swapped_window = false;
            }

            if (input.get_key(GLFW_KEY_SPACE)) {
                if ((held % 5) == 0) {
                    Vec3 p = c_entity.get<TransformComponent>().position;
                    Vec3 v = c_entity.get<TransformComponent>().forward();
                    octoswarm.set_position(chambered_cow, p + 3 * v, v);

                    chambered_cow = (chambered_cow + 1) % octoswarm.n_cows;
                }
                held++;
            } else {
                held = 0;
            }

            double frame_time = 0.0;
            double this_time = last_time;
            while (frame_time < 0.016667) {
                this_time = glfwGetTime();
                frame_time = this_time - last_time;
                std::this_thread::yield();
            }
            last_time = this_time;

            framerates.erase(framerates.begin());
            framerates.push_back(1.0 / frame_time);

            frame_avg += frame_time;

            if ((frame % 10) == 0) {
                frametime = 1000 * frame_avg / 10.0;
                framerate = 0.5 + 10.0 / frame_avg;
                frame_avg = 0;
            }

            if (simulating || single_step) {
                double sim_time =
                    sim_scene.step(playback / 60.f, it_count, true);

                sim_frame_avg += sim_time;

                if ((sim_frame % 10) == 0) {
                    sim_frametime = sim_frame_avg / 10.0;
                    sim_frame_avg = 0;
                }

                single_step = false;

                sim_frame++;
            }

            scene.update();

            frame++;
        });

    app.run();

    return 0;
}